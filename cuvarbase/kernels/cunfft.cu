#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>

#define RESTRICT __restrict__
#define CONSTANT const
#define PI 3.14159265358979323846264338327950288f
#define FILTER gauss_filter
//{CPP_DEFS}

#ifdef DOUBLE_PRECISION
	#define ATOMIC_ADD atomicAddDouble
	#define FLT double

#else
	#define ATOMIC_ADD atomicAdd
	#define FLT float
#endif

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ FLT gauss_filter(CONSTANT FLT x, CONSTANT FLT b) {
	return expf(-(x*x) / b) / sqrtf(PI * b);
}

__device__ int mod(CONSTANT int a, CONSTANT int b) {
   int ret = a % b;
   return (ret < 0) ? ret + b : ret;
}

__device__ FLT diffmod(CONSTANT FLT a, CONSTANT FLT b, CONSTANT FLT M) {
	FLT ret = a - b;
	if (fabsf(ret) > M/2){
		if (ret > 0)
			return ret - M;
		return M + ret;
	}
	return ret;
}

__global__ void center_fft(
	FLT * RESTRICT in,
	pycuda::complex<FLT> *out,
	CONSTANT int n,
	CONSTANT int nbatch){

	int i = blockIdx.x *blockDim.x + threadIdx.x;

	int batch = i / n;

	if (batch < nbatch) {
		int k = mod(i, n);

		int shift = (k % 2 == 0) ? 1 : -1;
		out[i] = pycuda::complex<FLT>(in[batch * n + k] * shift, 0.f);
	}
}

__global__ void precompute_psi(
	FLT *RESTRICT x, // observation times
	FLT * q1,        // precomputed filter values (length n0)
	FLT * q2,        // precomputed filter values (length n0)
	FLT * q3,        // precomputed filter values (length 2 * m + 1)
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int m,      // max filter radius
	CONSTANT FLT b)      // filter scaling
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	FLT binv = 1.f/b;
	if (i < n0){

		FLT xg = m + (n * x[i] - floorf(n * x[i]));

		q1[i] = expf(-xg * (xg * binv)) / sqrtf(b * PI);
		q2[i] = expf( 2.f * xg * binv);

	} else if (i - n0 < 2 * m + 1) {
		int l = i - n0;
		q3[l] = expf(-l * l * binv);
	}
}

__global__ void precompute_psi_noscale(
	FLT *RESTRICT x, // observation times
	FLT * q1,        // precomputed filter values (length n0)
	FLT * q2,        // precomputed filter values (length n0)
	FLT * q3,        // precomputed filter values (length 2 * m + 1)
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int m,      // max filter radius
	CONSTANT FLT b,      // filter scaling
	CONSTANT FLT x0,     // min(x)
	CONSTANT FLT xf,     // max(x)
	CONSTANT FLT spp)    // samples per peak
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	FLT binv = 1.f/b;
	if (i < n0){
		FLT xg = (x[i] - x0) / (spp * (xf - x0)) - 0.5f;

		xg = m + (n * xg - floorf(n * xg));

		q1[i] = expf(-xg * (xg * binv)) / sqrtf(b * PI);
		q2[i] = expf( 2.f * xg * binv);

	} else if (i - n0 < 2 * m + 1) {
		int l = i - n0;
		q3[l] = expf(-l * l * binv);
	}

}


__global__ void fast_gaussian_grid(
	FLT *RESTRICT x,     // data (observation times), length n0
	FLT *RESTRICT y,     // data (observations), length nbatch * n0
	FLT * grid,          // grid, length n * nbatch
	FLT *RESTRICT q1,	 // precomputed filter values
	FLT *RESTRICT q2,	 // precomputed filter values
	FLT *RESTRICT q3,	 // precomputed filter values
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int nbatch, // number of grids/datasets
	CONSTANT int m){     // max filter radius

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int batch = i / n0;

	if (batch < nbatch){
		// datapoint
		int di = i % n0;

		// observation
		FLT yi = y[i];

		// nearest gridpoint (rounding down)
		int u = (int) floorf(n * (x[di] + 0.5f) - m);

		// precomputed filter values
		FLT Q  = q1[di];
		FLT Q2 = q2[di];

		// add datapoint to grid
		for(int k = u; k < u + 2 * m + 1; k++){
			ATOMIC_ADD(grid + mod(k, n) + batch * n, Q * q3[k - u] * yi);
			Q *= Q2;
		}
	}
}

__global__ void fast_gaussian_grid_noscale(
	FLT *RESTRICT x,     // data (observation times), length n0
	FLT *RESTRICT y,     // data (observations), length nbatch * n0
	FLT * grid,          // grid, length n * nbatch
	FLT *RESTRICT q1,	 // precomputed filter values
	FLT *RESTRICT q2,	 // precomputed filter values
	FLT *RESTRICT q3,	 // precomputed filter values
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int nbatch, // number of grids/datasets
	CONSTANT int m,      // max filter radius
	CONSTANT FLT x0,     // min(x)
	CONSTANT FLT xf,     // max(x)
	CONSTANT FLT spp)    // samples per peak
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int batch = i / n0;

	if (batch < nbatch){

		// datapoint
		int di = i % n0;

		// scale
		FLT xval = (x[di] - x0) / (spp * (xf - x0)) - 0.5f;

		// observation
		FLT yi = y[i];

		// nearest gridpoint (rounding down)
		int u = (int) floorf(n * (xval + 0.5f) - m);

		// precomputed filter values
		FLT Q  = q1[di];
		FLT Q2 = q2[di];

		// add datapoint to grid
		for(int k = u; k < u + 2 * m + 1; k++){
			ATOMIC_ADD(grid + mod(k, n) + batch * n, Q * q3[k - u] * yi);
			Q *= Q2;
		}
	}
}



__global__ void slow_gaussian_grid(
	FLT *RESTRICT x,     // data (observation times)
	FLT *RESTRICT y,     // data (observations)
	FLT * grid,          // grid
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int nbatch, // number of grids
	CONSTANT int m,      // max filter radius
	CONSTANT FLT b)      // filter scaling
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int batch = i / n;

	if (batch < nbatch){
		FLT dx, dgi;

		// grid index for this thread
		int grid_index = i - n * batch;

		// iterate through data
		for(int di = 0; di < n0; di ++){

			// grid index of datapoint (float)
			dgi = n * (x[di] + 0.5f);

			// "distance" between grid_index and datapoint
			dx = diffmod(dgi, grid_index, n);

			// skip if datapoint too far away
			if (dx > m)
				continue;

			// add (weighted) datapoint to grid
			grid[i] += FILTER(dx, b) * y[di + n0 * batch];
		}
	}
}

__global__ void slow_gaussian_grid_noscale(
	FLT *RESTRICT x,     // data (observation times)
	FLT *RESTRICT y,     // data (observations)
	FLT * grid,          // grid
	CONSTANT int n0,     // data size
	CONSTANT int n,      // grid size
	CONSTANT int nbatch, // number of grids
	CONSTANT int m,      // max filter radius
	CONSTANT FLT b,      // filter scaling
	CONSTANT FLT x0,     // min(x)
	CONSTANT FLT xf,     // max(x)
	CONSTANT FLT spp)    // samples per peak
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int batch = i / n;

	if (batch < nbatch){
		FLT dx, dgi;



		// grid index for this thread
		int grid_index = i - n * batch;

		// iterate through data
		for(int di = 0; di < n0; di ++){

			// scale
			FLT xval = (x[di] - x0) / (spp * (xf - x0)) - 0.5f;

			// grid index of datapoint (float)
			dgi = n * (xval + 0.5f);

			// "distance" between grid_index and datapoint
			dx = diffmod(dgi, grid_index, n);

			// skip if datapoint too far away
			if (dx > m)
				continue;

			// add (weighted) datapoint to grid
			grid[i] += FILTER(dx, b) * y[di + n0 * batch];
		}
	}
}

__global__ void divide_phi_hat(
	pycuda::complex<FLT> *gin,
	pycuda::complex<FLT> *gout,
	CONSTANT int n, // sigma * N
	CONSTANT int N, // number of desired frequency samples
	CONSTANT int nbatch, // number of transforms
	CONSTANT FLT b,    // scale factor
	CONSTANT FLT phi0) // (unscaled) phase shift resulting from t[0] != 0
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	int batch = i / N;

	pycuda::complex<FLT> I = pycuda::complex<FLT>(0.f, 1.f);

	if (batch < nbatch){
		int m = i % N;

		int kprime = m - N/2;
		FLT Kprime = (PI * kprime) / n;
		int k = m + (n-N)/2;

		pycuda::complex<FLT> G = gin[batch * n + k];

		// *= exp(i * (2 * pi * phi0) * (k - n / 2)) for t[0] != 0
		FLT theta_k = 2.f * PI * phi0 * kprime;
		G *= pycuda::complex<FLT>(cosf(theta_k), sinf(theta_k));

		// Not sure why this is needed but necessary to be consistent
		// with jake vanderplas' NFFT (and I assume any other implementation)
		G *= (m % 2 == 0) ? 1.f : -1.f;

		// normalization factor from gridding kernel (gaussian)
		gout[i] = G * exp(b * Kprime * Kprime);
	}

}


__global__ void divide_phi_hat_noscale(
	pycuda::complex<FLT> *gin,
	pycuda::complex<FLT> *gout,
	CONSTANT int n, // sigma * N
	CONSTANT int N, // number of desired frequency samples
	CONSTANT int nbatch, // number of transforms
	CONSTANT FLT b,      // filter scaling
	CONSTANT FLT x0,     // min(x)
	CONSTANT FLT xf,     // max(x)
	CONSTANT FLT spp)    // samples per peak
{
	int i = blockIdx.x *blockDim.x + threadIdx.x;

	int batch = i / N;

	pycuda::complex<FLT> I = pycuda::complex<FLT>(0.f, 1.f);

	if (batch < nbatch){
		int m = i % N;

		int kprime = m - N/2;
		FLT Kprime = (PI * kprime) / n;
		int k = m + (n-N)/2;

		pycuda::complex<FLT> G = gin[batch * n + k];

		// *= exp(i * (2 * pi * phi0) * (k - n / 2)) for t[0] != 0
		FLT phi0 = x0 / (spp * (xf - x0));
		FLT theta_k = 2.f * PI * phi0 * kprime;
		G *= pycuda::complex<FLT>(cosf(theta_k), sinf(theta_k));

		// Not sure why this is needed but necessary to be consistent
		// with jake vanderplas' NFFT (and I assume any other implementation)
		//G *= (m % 2 == 0) ? 1.f : -1.f;

		// normalization factor from gridding kernel (gaussian)
		gout[i] = G * exp(b * Kprime * Kprime);
	}

}

